#include "cudaRelated.h"
#include <cutil.h>
#include <cutil_inline.h>
#include "cudaKernel.cu"
#include <iostream>
using namespace std;

void memAlloc(void ** address, int size) {
	hipHostAlloc(address, size, hipHostMallocDefault);
}

void memDealloc(void * address) {
	if (address != 0)
		hipHostFree(address);
}

void cudaMemAlloc(void **address, int size,bool isMemset) {
	hipMalloc(address, size);
	if(isMemset == true){
		hipMemset(address, 0, size);
	}
}
void cudaMemFree(void * address) {
	if (address != 0)
		hipFree(address);
}

void cudaMemoryCopy(void * h_address, void * d_address,int size){
	hipMemcpy(d_address, h_address,size, hipMemcpyHostToDevice);
}

void cudaDistance(cudaGrid * h_cudaArray, cudaGrid * d_cudeArray, cudaGrid * h_curGrid, cudaGrid * d_curGrid, int size, int * d_result, int * h_result,int type,int * h_pointCnt,int * d_pointCnt) {
	unsigned int timer = 0;
	cutCreateTimer(&timer);
	cutStartTimer(timer);
	hipMemset(d_result,0,sizeof(int)*size);
	expand<<<size/MAXTHREAD,MAXTHREAD>>>(d_cudeArray,d_curGrid,d_result,type,d_pointCnt,size,eps,DIMENSION,minPts);
	hipMemcpy(h_pointCnt,d_pointCnt,sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(h_result, d_result, sizeof(int)*size, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	cutStopTimer(timer);
	//cout << "timer =  " << cutGetTimerValue(timer) << endl;
}

void cudaMap(cudaGrid * hipArray, int localGridNum, int * gridnum, int * stride1d, int * stride2d, int * h_intermediate)//h_intermediate will be allocated outside.
{
	unsigned int timer = 0;
	for (int i = 0; i < 100; i++) {
		cout << h_intermediate[i] << ",";
	}
	cout << endl;
	int globalGridNum = stride1d[localGridNum - 1] + gridnum[localGridNum - 1];
	int globalGridNum2 = stride2d[localGridNum - 1] + gridnum[localGridNum - 1] * gridnum[localGridNum - 1];
	cutCreateTimer(&timer);
	cutStartTimer(timer);

	cudaGrid * d_cudaGrid = 0;
	hipMalloc((void**) &d_cudaGrid, sizeof(cudaGrid) * globalGridNum);
	hipMemset(d_cudaGrid, 0, sizeof(cudaGrid) * globalGridNum);
	hipMemcpy(d_cudaGrid, hipArray, sizeof(cudaGrid) * globalGridNum, hipMemcpyHostToDevice);

	int * d_intermediate = 0;
	hipMalloc((void**) &d_intermediate, sizeof(int) * globalGridNum2);
	hipMemset(d_intermediate, 0, sizeof(int) * globalGridNum2);

	int * d_gridnum = 0;
	hipMalloc((void**) &d_gridnum, sizeof(int) * localGridNum);
	hipMemcpy(d_gridnum, gridnum, sizeof(int) * localGridNum, hipMemcpyHostToDevice);

	int * d_stride1d = 0;
	hipMalloc((void**) &d_stride1d, sizeof(int) * localGridNum);
	hipMemcpy(d_stride1d, stride1d, sizeof(int) * localGridNum, hipMemcpyHostToDevice);

	int * d_stride2d = 0;
	hipMalloc((void**) &d_stride2d, sizeof(int) * localGridNum);
	hipMemcpy(d_stride2d, stride2d, sizeof(int) * localGridNum, hipMemcpyHostToDevice);

	hipDeviceSynchronize();
	dim3 blocksize(localGridNum, MAXTHREAD);

	mapGrid<<<blocksize,MAXTHREAD>>>(d_cudaGrid,d_intermediate,d_gridnum,d_stride1d,d_stride2d);
	hipDeviceSynchronize();

	hipMemcpy(h_intermediate, d_intermediate, sizeof(int) * globalGridNum2, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	cutStopTimer(timer);
	cout << "timer =  " << cutGetTimerValue(timer) << endl;

	hipFree(d_cudaGrid);
	hipFree(d_intermediate);
	hipFree(d_gridnum);
	hipFree(d_stride1d);
	hipFree(d_stride2d);
}

