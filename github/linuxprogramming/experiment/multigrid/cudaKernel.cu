#include "hip/hip_runtime.h"
#ifndef CUDAKERNEL
#define CUDAKERNEL

#include "cudaRelated.h"

__global__ void mapGrid(cudaGrid * gridArray, int * intermediate, int * gridnum, int * stride1d, int * stride2d) {
	int blockidx = blockIdx.x;
	int blockidy = blockIdx.y;
	int threadid = threadIdx.x;
	int roundtime = 1;
	float distance = 0;
	float temp = 0;
	cudaGrid * offsetAddress = gridArray + stride1d[blockidx];
	cudaGrid * curColGrid = 0;
	cudaGrid * curRowGrid = 0;
	int blockOffset = 0;
	int threadOffset = 0;
	if (blockidy < gridnum[blockidx] && threadid < gridnum[blockidx]) {
		if (gridnum[blockidx] > MAXTHREAD) {
			roundtime = gridnum[blockidx] / MAXTHREAD + 1;
		}
		for (int k = 0; k < roundtime; ++k) {
			blockOffset = blockidy + k * MAXTHREAD;
			curRowGrid = offsetAddress + blockOffset;
			for (int j = 0; j < roundtime; ++j) {
				threadOffset = threadid + j * MAXTHREAD;
				curColGrid = offsetAddress + threadOffset;
				distance = 0;
				for (int i = 0; i < DIMENSION_CUDA; ++i) {
					temp = powf(curRowGrid->cf[i] - curColGrid->cf[i], 2);
					distance += temp;
				}
				if (distance > 0) {
					//					intermediate[1]=1;
					intermediate[stride2d[blockidx] + blockOffset * gridnum[blockidx] + threadOffset] = 1;
				}
			}
		}
	}
}

__global__ void reduceGrid(int * intermediate, int * gridnum) {

}


__device__ bool strcmp(char * str1,char * str2){
	while(*str1 && *str2 && *str1++ == *str2++);
	if(*str1 == '\0' && *str2 == '\0')return true;
	else return false;
}

__global__ void expand(cudaGrid * hipArray,cudaGrid * curGrid, int * result,int type,int * d_pointCnt,int size,float eps,int dimension,int minpts){
	int threadid = threadIdx.x +blockIdx.x* MAXTHREAD;
	if(threadid >= size){
		return;
	}
	cudaGrid * grid = hipArray + threadid;
	if(strcmp(curGrid->globalGridKey,grid->globalGridKey))return ;
	if(grid->status == SPARSE)return;
	if(type == FINDNEIGHBOR && grid->clusterID > UNCLASSIFIED && *d_pointCnt >= minpts)return;
	if(type == INCREMENTALFINDNEIGHBOR && grid->clusterID == curGrid->clusterID)return;
	float distance = 0;
	for(int i =0 ;i < dimension;++i){
		distance += powf(grid->cf[i] - curGrid->cf[i],2);
	}
	if(distance < eps *eps/* && distance !=0*/){
		result[threadid] = 1;
		atomicAdd(d_pointCnt,1);
	}

}

#endif

