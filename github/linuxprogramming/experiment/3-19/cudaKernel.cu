#include "hip/hip_runtime.h"
#ifndef CUDAKERNEL
#define CUDAKERNEL

#include "cudaRelated.h"

__global__ void mapGrid(cudaGrid * gridArray, int * intermediate, int * gridnum, int * stride1d, int * stride2d) {
	int blockidx = blockIdx.x;
	int blockidy = blockIdx.y;
	int threadid = threadIdx.x;
	int roundtime = 0;
	float distance = 0;
	cudaGrid * offsetAddress = gridArray + stride1d[blockidx];
//	cudaGrid * curColGrid = 0;
//	cudaGrid * curRowGrid  =0;
//	int blockOffset = 0;
//	int threadOffset = 0;
//	//intermediate[0] = 100;
//	if (blockidy < gridnum[blockidx] && threadid < gridnum[blockidx]) {
//		if (gridnum[blockidx] > MAXTHREAD) {
//			roundtime = gridnum[blockidx] / MAXTHREAD + 1;
//		}
//		for (int k = 0; k < roundtime; ++k) {
//			blockOffset = blockidy + k * MAXTHREAD;
//			curRowGrid =offsetAddress + blockOffset ;
//			for (int j = 0; j < roundtime; ++j) {
//				threadOffset = threadid + j * MAXTHREAD;
//				curColGrid = offsetAddress + threadOffset;
//				distance = 0;
//				for (int i = 0; i < DIMENSION_CUDA; ++i) {
//					distance += powf(curRowGrid->cf[i] - curColGrid->cf[i], 2);
//				}
//				intermediate[stride2d[blockidx] +blockOffset* gridnum[blockidx] + threadOffset ] = 1;
//
//				//			if ( distance > 0.0) {// sqrtf(distance) <= EPS_CUDA &&
//				//				intermediate[offset * MAXTHREAD + blockidy * MAXTHREAD + threadid]= 1;
//				//			}
//			}
//		}
//	}
}
__global__ void reduceGrid(int * intermediate, int * gridnum) {

}

#endif

