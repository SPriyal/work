#include "cudaRelated.h"
#include <cutil.h>
#include <cutil_inline.h>
#include "cudaKernel.cu"
#include <iostream>
using namespace std;

int* cudaAlloc(int size) {
	int * address = 0;
	hipHostAlloc((void**) &address, size, hipHostMallocDefault);
	return address;
}

void cudaDealloc(int * address) {
	hipHostFree(address);
}

void cudaMap(cudaGrid * hipArray, int localGridNum, int * gridnum, int * stride1d, int * stride2d, int * h_intermediate)//h_intermediate will be allocated outside.
{
	unsigned int timer = 0;
	cout << h_intermediate[0] << endl;
	cutCreateTimer(&timer);
	cutStartTimer(timer);
	int globalGridNum = stride1d[localGridNum - 1] + gridnum[localGridNum - 1];
	int globalGridNum2 = stride2d[localGridNum - 1] + gridnum[localGridNum - 1] * gridnum[localGridNum - 1];

	cudaGrid * d_cudaGrid = 0;
	hipMalloc((void**) &d_cudaGrid, sizeof(cudaGrid) * globalGridNum);
	hipMemcpy(d_cudaGrid, hipArray, sizeof(cudaGrid) * globalGridNum, hipMemcpyHostToDevice);
	int * d_intermediate = 0;
	hipMalloc((void**) &d_intermediate, sizeof(int) * globalGridNum2);
	int * d_gridnum = 0;
	hipMalloc((void**) &d_gridnum, sizeof(int) * localGridNum);
	hipMemcpy(d_gridnum, &gridnum, sizeof(int), hipMemcpyHostToDevice);
	int * d_stride1d = 0;
	hipMalloc((void**) &d_stride1d, sizeof(int) * localGridNum);
	hipMemcpy(d_stride1d, stride1d, sizeof(int) * localGridNum, hipMemcpyHostToDevice);
	
	int * d_stride2d = 0;
	hipMalloc((void**) &d_stride2d, sizeof(int) * localGridNum);
	hipMemcpy(d_stride1d, stride2d, sizeof(int) * localGridNum, hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	dim3 blocksize(localGridNum, MAXTHREAD);

	mapGrid<<<blocksize,MAXTHREAD>>>(hipArray,d_intermediate,d_gridnum,stride1d,stride2d);
	hipDeviceSynchronize();

	hipMemcpy(h_intermediate, d_intermediate, sizeof(int) * globalGridNum2, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	cutStopTimer(timer);
	cout << "timer =  " << cutGetTimerValue(timer) << endl;

	cout << h_intermediate[0] << endl;
	//		for(int i = 0;i<localGridNum;++i){
	//			cout<<"localgrid "<<i<<endl;
	//			for(int j = 0;j< MAXTHREAD;++j){
	//				cout<<"line "<<j<<endl;
	//				for(int k = 0;k<MAXTHREAD;++k){
	//					cout<<h_intermediate[i * MAXTHREAD * MAXTHREAD + j * MAXTHREAD + k]<<", ";
	//				}
	//				cout<<endl;
	//			}
	//		}

	hipFree(d_cudaGrid);
	hipFree(d_intermediate);
	hipFree(d_gridnum);
	hipFree(d_stride1d);
	hipFree(d_stride2d);
}

