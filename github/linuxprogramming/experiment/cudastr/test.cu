#include "hip/hip_runtime.h"
#ifndef TEST
#define TEST
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cutil.h>
#include <cutil_inline.h>
using namespace std;

//typedef __device__ char * (*func)(char *dest, const char *src, int size);

__device__ char *StrCpy(char *dest, const char *src, int size)
{
	char4 *dest1 = (char4*)dest;
	const char4 *src1 = (char4*)src;

	int numChar4=size>>2;
	int i=0;
	for(i=0;i<numChar4;i++)
	{
		dest1[i]=src1[i];
	}
	for(i=(numChar4<<2);i<size;i++)
	{
		dest[i]=src[i];
	}
	return dest; 
}


__device__ char *StrCpy1(char *dest, const char *src, int size)
{
	char4 *dest1 = (char4*)dest;
	const char4 *src1 = (char4*)src;

	for(int i = 0 ; i < size;++i){
		dest[i] = src[i];
	}
	return dest; 
}

__global__ void testStrcpy1(char * str1 , char * str2 , int size){
	StrCpy1(str1,str2,size);
}

__global__ void testStrcpy(char * str1 , char * str2 , int size){
	StrCpy(str1,str2,size);
}




void callCuda(){
	const int SIZE =(1<<24);
	unsigned int timer=0;
	unsigned int timer2 = 0;
	cutilCheckError(cutCreateTimer(&timer));
	cutilCheckError(cutCreateTimer(&timer2));
	char * str1 = (char *)malloc(SIZE);
	memset(str1,0,SIZE);
	char * d_str1 = NULL;
	cutilSafeCall(hipMalloc((void**)&d_str1,SIZE));
	cutilSafeCall(hipMemset(d_str1,0,SIZE));
	cutilSafeCall(hipMemcpy(d_str1,str1,SIZE,hipMemcpyHostToDevice));
	char * d_str2 = NULL;
	cutilSafeCall(hipMalloc((void**)&d_str2,SIZE));
	cutilSafeCall(hipMemset(d_str2,0,SIZE));
	dim3 block(256,256);
	cutilCheckError(cutStartTimer(timer));
	testStrcpy<<<1,1>>>(d_str2,d_str1,SIZE);
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckError(cutStopTimer(timer));

	cutilCheckError(cutStartTimer(timer2));
	testStrcpy1<<<1,1>>>(d_str2,d_str1,SIZE);
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckError(cutStopTimer(timer2));

	printf("timer 1 = %f\n",cutGetTimerValue(timer));/**/
	printf("timer 2 = %f",cutGetTimerValue(timer2));/**/

	cutilCheckError(cutDeleteTimer(timer));
	cutilCheckError(cutDeleteTimer(timer2));
	cutilSafeCall(hipFree(d_str1));
	cutilSafeCall(hipFree(d_str2));
	hipDeviceReset();

}
#endif

/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation and 
 * any modifications thereto.  Any use, reproduction, disclosure, or distribution 
 * of this software and related documentation without an express license 
 * agreement from NVIDIA Corporation is strictly prohibited.
 * 
 *//*

#ifndef _CLOCK_KERNEL_H_
#define _CLOCK_KERNEL_H_

// This kernel computes a standard parallel reduction and evaluates the
// time it takes to do that for each block. The timing results are stored 
// in device memory.
__global__ static void timedReduction(const float * input, float * output, clock_t * timer)
{
    // __shared__ float shared[2 * blockDim.x];
    extern __shared__ float shared[];

    const int tid = threadIdx.x;
    const int bid = blockIdx.x;

    if (tid == 0) timer[bid] = clock();

    // Copy input.
    shared[tid] = input[tid];
    shared[tid + blockDim.x] = input[tid + blockDim.x];

    // Perform reduction to find minimum.
    for(int d = blockDim.x; d > 0; d /= 2)
    {
        __syncthreads();

        if (tid < d)
        {
            float f0 = shared[tid];
            float f1 = shared[tid + d];
            
            if (f1 < f0) {
                shared[tid] = f1;
            }
        }
    }

    // Write result.
    if (tid == 0) output[bid] = shared[0];

    __syncthreads();

    if (tid == 0) timer[bid+gridDim.x] = clock();
}

#endif // _CLOCK_KERNEL_H_
*/